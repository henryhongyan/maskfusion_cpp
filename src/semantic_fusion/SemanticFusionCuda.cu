#include "hip/hip_runtime.h"
/*
 * This file is part of SemanticFusion.
 *
 * Copyright (C) 2017 Imperial College London
 * 
 * The use of the code within this file and all code within files that 
 * make up the software that is SemanticFusion is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/semantic-fusion/semantic-fusion-license/> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 */

#include <iostream>
#include <vector>
#include <stdio.h>
#include <assert.h> 

#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaarithm.hpp>

#include <hip/hip_runtime.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool
        abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    } 
}

__global__ 
void semanticTableUpdate(hipTextureObject_t ids, const float *scores, const float *class_ids,
                             const int ids_width, const int ids_height, const int prob_channels,
			     float* map_table, float* map_max, const int map_size) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int surfel_id = tex2D<int>(ids,x,y);
    // New uniqueness code
    const int check_patch = 16;
    const int x_min = (x - check_patch) < 0 ? 0 : (x - check_patch);
    const int x_max = (x + check_patch) > 640 ? 640 : (x + check_patch);
    const int y_min = (y - check_patch) < 0 ? 0 : (y - check_patch);
    const int y_max = (y + check_patch) > 480 ? 480 : (y + check_patch);
    int first_h, first_w;
    for (int h = y_min; h < y_max; ++h) {
        for (int w = x_min; w < x_max; ++w) {
            int other_surfel_id = tex2D<int>(ids,w,h);
            if (other_surfel_id == surfel_id) {
                first_h = h;
                first_w = w;
                break;
            }
        }
    }
    if (first_h != y || first_w != x) {
        surfel_id = 0;
    }
    if (surfel_id > 0) {
        const float* probability = scores + (y * ids_width + x);
        float* prior_probability = map_table + surfel_id;
        float total = 0.0;
	
        for (unsigned char class_id = 0; class_id < prob_channels; ++class_id) {
          // if pixel (x, y) is not inferred as background by MaskRCNN
          if (fabsf(class_ids[y * ids_width + x]) >= 1e-5) {
	    if (fabsf(class_ids[y * ids_width + x] - class_id) <= 1e-5) {
	      prior_probability[0] *= probability[0];
	    }
	    else {
	      prior_probability[0] *= ((1.0f - probability[0])/(prob_channels - 1));
	    }
	  }
	  else {
	    prior_probability[0] *= (1.0f / prob_channels);
	  }
            total += prior_probability[0];
            prior_probability += map_size;
        }
        // Reset the pointers to the beginning again
        prior_probability = map_table + surfel_id;
        float max_probability = 0.0;
        int max_class = -1;
        float new_total = 0.0;
        for (int class_id = 0; class_id < prob_channels; ++class_id) {
            // Something has gone unexpectedly wrong - reinitialse
            if (total <= 1e-5) {
                prior_probability[0] = 1.0f / prob_channels;
            } else {
                prior_probability[0] /= total;
                if (prior_probability[0] > max_probability) {
                    max_probability = prior_probability[0];
                    max_class = class_id;
                }
            }
            new_total += prior_probability[0];
            prior_probability += map_size;
        }
        map_max[surfel_id] = static_cast<float>(max_class);
        map_max[surfel_id + map_size] = max_probability;
        map_max[surfel_id + map_size + map_size] += 1.0;
    }
}

__host__ 
void fuseSemanticProbabilities(hipTextureObject_t ids, const float *scores, const float *class_ids,
                             const int ids_width, const int ids_height, const int prob_channels,
			     float* map_table, float* map_max, const int map_size) {
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32;
    dim3 dimGrid(blocks,blocks);
    dim3 dimBlock(640/blocks,480/blocks);
    semanticTableUpdate<<<dimGrid,dimBlock>>>(ids,scores,class_ids,ids_width,ids_height,prob_channels,map_table,map_max,map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__
void updateTable(int n, const int* deleted_ids, const int num_deleted, const int current_table_size,
                 const int prob_width, const int prob_height, const int new_prob_width,
		 float const* probability_table, float* new_probability_table, float const * map_table, float* new_map_table) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        const int class_id = index / new_prob_width;
        const int component_id = index - (class_id * new_prob_width);
        const int new_id = (class_id * prob_width) + component_id;
        if (component_id >= num_deleted) {
            // Initialise to prior (prob height is the number of classes)
            new_probability_table[new_id] = 1.0f / prob_height;
            // Reset the max class surfel colouring lookup
            new_map_table[component_id] = -1.0;
            new_map_table[component_id + prob_width] = -1.0;
            new_map_table[component_id + prob_width + prob_width] = 0.0;
        } else {
            int offset = deleted_ids[component_id];
            new_probability_table[new_id] = probability_table[(class_id * prob_width) + offset];
            // Also must update our max class mapping
            new_map_table[component_id] = map_table[offset];
            new_map_table[component_id + prob_width] = map_table[prob_width + offset];
            new_map_table[component_id + prob_width + prob_width] = map_table[prob_width + prob_width + offset];
        }
    }
}

__host__
void updateProbabilityTable(int* filtered_ids, const int num_filtered, const int current_table_size,
                               const int prob_width, const int prob_height, const int new_prob_width,
			       float const* probability_table, float* new_probability_table,
                               float const* map_table, float* new_map_table) {
    const int threads = 512;
    const int num_to_update = new_prob_width * prob_height;
    const int blocks = (num_to_update + threads - 1) / threads;
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateTable<<<dimGrid,dimBlock>>>(num_to_update,filtered_ids,num_filtered,current_table_size,prob_width,prob_height,new_prob_width,probability_table,new_probability_table,map_table,new_map_table);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__ 
void renderProbabilityMapKernel(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probability_table, const int prob_width, const int prob_height, 
                          float* rendered_probabilities) 
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int surfel_id = tex2D<int>(ids,x,y);
    int projected_probability_offset = y * ids_width + x;
    int probability_table_offset = surfel_id;
    for (int class_id = 0; class_id < prob_height; ++class_id) {
        if (surfel_id > 0) {
            rendered_probabilities[projected_probability_offset] = probability_table[probability_table_offset];
        } else {
            rendered_probabilities[projected_probability_offset] = ((class_id == 0) ? 1.0 : 0.0);
        }
        projected_probability_offset += (ids_width * ids_height);
        probability_table_offset += prob_width;
    }
}

__host__
void renderProbabilityMap(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probability_table, const int prob_width, const int prob_height, 
                          float* rendered_probabilities) 
{
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32;
    dim3 dimGrid(blocks,blocks);
    dim3 dimBlock(ids_width/blocks,ids_height/blocks);
    renderProbabilityMapKernel<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,probability_table,prob_width,prob_height,rendered_probabilities);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__ 
void updateMaxClassKernel(const int n, const float* probabilities, const int classes,
                          float* map_max, const int map_size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        // Reset the pointers to the beginning again
        const float* probability = probabilities + index;
        probability += map_size;
        float max_probability = 0.0;
        int max_class = -1;
        for (int class_id = 1; class_id < classes; ++class_id) {
            if (probability[0] > max_probability) {
                max_probability = probability[0];
                max_class = class_id;
            }
            probability += map_size;
        }
        map_max[index] = static_cast<float>(max_class);
        map_max[index + map_size] = max_probability;
    }
}

__host__ 
void updateMaxClass(const int n, const float* probabilities, const int classes,
                    float* map_max, const int map_size)
{
    const int threads = 512;
    const int blocks = (n + threads - 1) / threads;
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateMaxClassKernel<<<dimGrid,dimBlock>>>(n,probabilities,classes,map_max,map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__
void showSurfelIdKernel(hipTextureObject_t ids) {
  int surfel_id = tex2D<int>(ids,320,240);
  if (tex2D<int>(ids,320,240)) {
    printf("********************** x = 320 && y = 240 : %d \n", surfel_id);
  }
}

__host__
void showSurfelId(hipTextureObject_t ids) {
  dim3 dimGrid(1);
  dim3 dimBlock(1);
  showSurfelIdKernel<<<dimGrid, dimBlock>>>(ids);
  gpuErrChk(hipGetLastError());
  gpuErrChk(hipDeviceSynchronize());
}